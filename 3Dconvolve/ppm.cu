#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "ppmKernel.cu"
#include "ppm.h"

// typedef struct {
//      unsigned char red,green,blue;
// } PPMPixel;

// typedef struct {
//      int x, y;
//      PPMPixel *data;
// } PPMImage;

#define CREATOR "RPFELGUEIRAS"
#define RGB_COMPONENT_COLOR 255

#define OUTPUT_TILE_SIZE 12


#define FATAL(msg, ...) \
    do {\
        fprintf(stderr, "[%s:%d] "msg"\n", __FILE__, __LINE__, ##__VA_ARGS__);\
        exit(-1);\
    } while(0)

static PPMImage *readPPM(const char *filename)
{
         char buff[16];
         PPMImage *img;
         FILE *fp;
         int c, rgb_comp_color;
         //open PPM file for reading
         fp = fopen(filename, "rb");
         if (!fp) {
              fprintf(stderr, "Unable to open file '%s'\n", filename);
              exit(1);
         }

         //read image format
         if (!fgets(buff, sizeof(buff), fp)) {
              perror(filename);
              exit(1);
         }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}
void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    //write the header file
    //image format
    fprintf(fp, "P6\n");

    //comments
    fprintf(fp, "# Created by %s\n",CREATOR);

    //image size
    fprintf(fp, "%d %d\n",img->x,img->y);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}

Filter3D * initializeFilter()
{
    int data[FILTER_SIZE][FILTER_SIZE][FILTER_SIZE] =  { { {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 1, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0} },
                                                         { {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 1, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0} },
                                                         { {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 1, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0} },
                                                         { {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 1, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0} },
                                                         { {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 1, 0, 0},
                                                           {0, 0, 0, 0, 0},
                                                           {0, 0, 0, 0, 0} }
                                                       };
    Filter3D * filter = (Filter3D*) malloc(sizeof(Filter3D));
    filter->x = FILTER_SIZE;
    filter->y = FILTER_SIZE;
    filter->z = FILTER_SIZE;
    for (int z = 0; z < FILTER_SIZE; z++)
        for (int y = 0; y < FILTER_SIZE; y++)
            for (int x = 0; x < FILTER_SIZE; x++)
                filter[z][y][x] = data[z][y][x];

    filter->factor = 1.0;
    filter->bias =0;
    return filter;
}

int main(){


    clock_t begin, end;
    double time_spent;


    /* here, do your time-consuming job */

    char instr[80];
    char outstr[80];
    int i = 0;

    PPMImage images[301];

    // for(i = 0; i < 301; i++) {
    //     sprintf(instr, "infiles/baby001.ppm", i+1);
    //     images[i] = *readPPM(instr);
    // }

    PPMPixel *imageData_d, *outputData_d, *outputData_h;
    Filter3D * filter_h = initializeFilter();

    hipError_t cuda_ret;

    for(i = 0; i < 301; i++) {
        sprintf(instr, "infiles/baby%03d.ppm", i+1);
        images[i] = *readPPM(instr);
    }

    PPMImage *image;
    image = &images[0];
    outputData_h = (PPMPixel *)malloc(image->x*image->y*sizeof(PPMPixel));

    cuda_ret = hipMalloc((void**)&(imageData_d), image->x*image->y*sizeof(PPMPixel));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&(outputData_d), image->x*image->y*sizeof(PPMPixel));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    PPMImage *outImage;
    outImage = (PPMImage *)malloc(sizeof(PPMImage));
    outImage->x = image->x;
    outImage->y = image->y;

    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter_h, sizeof(Filter));
    hipDeviceSynchronize();


    begin = clock();

    for(i = 0; i < 301; i++) {
        sprintf(outstr, "outfiles/baby%03d.ppm", i+1);

        image = &images[i];

        cuda_ret = hipMemcpy(imageData_d, image->data, image->x*image->y*sizeof(PPMPixel), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy to device");

        // Convolution
        const unsigned int grid_x = (image->x - 1) / OUTPUT_TILE_SIZE + 1;
        const unsigned int grid_y = (image->y -1) / OUTPUT_TILE_SIZE + 1;
        dim3 dim_grid(grid_x, grid_y, 1);
        dim3 dim_block(INPUT_TILE_SIZE, INPUT_TILE_SIZE, 1);
        convolution<<<dim_grid, dim_block>>>(imageData_d, outputData_d, image->x, image->y);

        cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

        cuda_ret = hipMemcpy(outputData_h, outputData_d, image->x*image->y*sizeof(PPMPixel), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy to host");


        outImage->data = outputData_h;

        writePPM(outstr,outImage);

    }
    end = clock();

    free(outputData_h);
    free(outImage);
    hipFree(imageData_d);
    hipFree(outputData_d);

    // for(i = 1; i <= 1; i++) {
        // sprintf(instr, "infiles/baby001.ppm");
        // sprintf(outstr, "outfiles/baby001.ppm");

        // PPMImage *image;
        // sprintf(instr, "infiles/baby001.ppm");

        // image = readPPM(instr);



        // changeColorPPM(&images[i-1]);


    // }

    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    printf("%f seconds spent\n", time_spent);

}