#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <time.h>
#include "ppmKernel.cu"
#include "ppm.h"

#define FATAL(msg, ...) \
    do {\
        fprintf(stderr, "[%s:%d] "msg"\n", __FILE__, __LINE__, ##__VA_ARGS__);\
        exit(-1);\
    } while(0)

static PPMImage *readPPM(const char *filename)
{
         char buff[16];
         PPMImage *img;
         FILE *fp;
         int c, rgb_comp_color;
         //open PPM file for reading
         fp = fopen(filename, "rb");
         if (!fp) {
              fprintf(stderr, "Unable to open file '%s'\n", filename);
              exit(1);
         }

         //read image format
         if (!fgets(buff, sizeof(buff), fp)) {
              perror(filename);
              exit(1);
         }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    //write the header file
    //image format
    fprintf(fp, "P6\n");

    //comments
    fprintf(fp, "# Created by %s\n",CREATOR);

    //image size
    fprintf(fp, "%d %d\n",img->x,img->y);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}

// Mallocs space for a filter, initializes it, and returns a pointer to it
Filter3D * initializeFilter()
{
    double data[FILTER_SIZE][FILTER_SIZE][FILTER_SIZE] =  { { {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0} },

                                                             { {0, 0, 0, 0, 0},
                                                               {0, -1, -1, -1, 0},
                                                               {0, -1, 8, -1, 0},
                                                               {0, -1, -1, -1, 0},
                                                               {0, 0, 0, 0, 0} },

                                                             { {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 4, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0} },

                                                             { {0, 0, 0, 0, 0},
                                                               {0, -1, -1, -1, 0},
                                                               {0, -1, 8, -1, 0},
                                                               {0, -1, -1, -1, 0},
                                                               {0, 0, 0, 0, 0} },
                                                             { {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0},
                                                               {0, 0, 0, 0, 0} }
                                                       };

    Filter3D * filter = (Filter3D*) malloc(sizeof(Filter3D));
    // Set filter dimensions
    filter->x = FILTER_SIZE;
    filter->y = FILTER_SIZE;
    filter->z = FILTER_SIZE;
    // Set filter data elements
    for (int z = 0; z < FILTER_SIZE; z++)
        for (int y = 0; y < FILTER_SIZE; y++)
            for (int x = 0; x < FILTER_SIZE; x++) {
                (filter->data)[z][y][x] = data[z][y][x];
            }
    // Set filter factor and bias
    filter->factor = .25;
    filter->bias =0;
    return filter;
}

// Given a z dimension within the video loads in the next INPUT_TILE_Z number
// of frames from the infiles directory
void loadFrames(PPMImage * frames, int z, int totalFrames)
{
    char instr[80];
    // Read in INPUT_TILE_Z number of frames
    for (int i = 0; i < INPUT_TILE_Z; i++)
    {
        int fileNum = i + z + 1 - FILTER_SIZE / 2;
        // Only read in frames that exist
        if (fileNum <= totalFrames && fileNum > 0)
        {
            sprintf(instr, "../infiles/tmp%03d.ppm", fileNum);
            frames[i] = *readPPM(instr);
        }
    }
}

// Given an x, y, and z dimension within the video gets a chunk of pixels from frames to be processed
// by the next kernel call
void getPixels(PPMImage frames[], PPMPixel *data, int x, int y, int z, int width, int height, int depth)
{
    for (int k = 0; k < INPUT_TILE_Z; k++)
    {
        for (int j = 0; j < INPUT_TILE_Y; j++)
        {
            for (int i = 0; i < INPUT_TILE_X; i++)
            {
                int data_x = i + x - FILTER_SIZE / 2;
                int data_y = j + y - FILTER_SIZE / 2;
                int data_z = k + z - FILTER_SIZE / 2;
                // if data is within bounds of the video file write the pixel data
                if ((data_x >= 0) && (data_x < width) && (data_y >= 0) && (data_y < height) &&
                    (data_z >= 0) && (data_z < depth))
                    data[k * INPUT_TILE_X * INPUT_TILE_Y + j * INPUT_TILE_X + i] = frames[k].data[data_y * width + data_x];
                // otherwise write a black pixel (all zeroes)
                else
                {
                    data[k * INPUT_TILE_X * INPUT_TILE_Y + j * INPUT_TILE_X + i].red = 0;
                    data[k * INPUT_TILE_X * INPUT_TILE_Y + j * INPUT_TILE_X + i].blue = 0;
                    data[k * INPUT_TILE_X * INPUT_TILE_Y + j * INPUT_TILE_X + i].green = 0;
                }
            }
        }
    }
}

// Given an x, y, anx z dimension within the video, writes the processed pixels out to memory
void writePixels(PPMPixel * data, PPMImage * frames, int x, int y, int z, int width, int height)
{
    for (int k = 0; k < OUTPUT_TILE_Z; k++)
        for (int j = 0; j < OUTPUT_TILE_Y; j++)
            for (int i = 0; i < OUTPUT_TILE_X; i++) {
                // if pixel is within bounds write it out to memory
                if(x+i < width && y + j < height)
                    frames[k].data[width*(y+j)+ x+i] = data[k * OUTPUT_TILE_X * OUTPUT_TILE_Y + i * OUTPUT_TILE_X + j];
            }
}

// Given a z dimension in the video, writes OUTPUT_TILE_Z number of frames to disk
void writeFrames(PPMImage * frames, int z, int totalFrames)
{
    char outstr[80];
    for (int i = 0; i < OUTPUT_TILE_Z; i++)
    {
        int fileNum = i + z + 1;
        // if frame is within bounds write it out to disk
        if (fileNum <= totalFrames)
        {
            sprintf(outstr, "../outfiles/tmp%03d.ppm", fileNum);
            writePPM(outstr, &(frames[i]));
        }
    }
}

int main(int argc, char *argv[]){
    char *infile = (char*)"foreman.mp4";
    char ffmpegString[200];

    // if argument is given, go to input_videos directory and create 301 frames of that video
    // in the infiles directory
    if(argc > 1) {
      infile = argv[1];
      if (!system(NULL)) {exit (EXIT_FAILURE);}
      system("exec rm -r ../infiles/*");
      sprintf(ffmpegString, "ffmpeg -i ../input_videos/%s -vframes 301 ../infiles/tmp%%03d.ppm", infile);
      system (ffmpegString);
    }

    // remove old frames from outfiles
    system("exec rm -r -f ../outfiles/*");

    int totalFrames = 0;
    DIR * dirp;
    struct dirent * entry;

    // get total number of frames that need to be processed
    dirp = opendir("../infiles");
    while ((entry = readdir(dirp)) != NULL) {
        if (entry->d_type == DT_REG) { /* If the entry is a regular file */
             totalFrames++;
        }
    }

    closedir(dirp);

    // Initialize profiling variables
    clock_t begin, end;
    double time_spent = 0.0;


    PPMPixel *imageData_d, *outputData_d, *outputData_h, *inputData_h;
    Filter3D * filter_h = initializeFilter();

    hipError_t cuda_ret;

    // Read first frame to get width and height information
    PPMImage *image =  readPPM("../infiles/tmp001.ppm");

    // Malloc space for input and output pixels
    inputData_h  = (PPMPixel *)malloc(INPUT_TILE_X * INPUT_TILE_Y * INPUT_TILE_Z * sizeof(PPMPixel));
    outputData_h = (PPMPixel *)malloc(OUTPUT_TILE_X * OUTPUT_TILE_Y * OUTPUT_TILE_Z * sizeof(PPMPixel));

    // Malloc space for input and output frames
    PPMImage inputFrames[INPUT_TILE_Z], outputFrames[OUTPUT_TILE_Z];
    for (int i = 0; i < INPUT_TILE_Z; i++) {
        inputFrames[i].x = image->x;
        inputFrames[i].y = image->y;
        inputFrames[i].data = (PPMPixel *)malloc(image->x * image->y * sizeof(PPMPixel));
    }
    for (int i = 0; i < OUTPUT_TILE_Z; i++) {
        outputFrames[i].x = image->x;
        outputFrames[i].y = image->y;
        outputFrames[i].data = (PPMPixel *)malloc(image->x * image->y * sizeof(PPMPixel));
    }

    // Malloc device memory for input and outputs
    cuda_ret = hipMalloc((void**)&(imageData_d), INPUT_TILE_X * INPUT_TILE_Y * INPUT_TILE_Z * sizeof(PPMPixel));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&(outputData_d), OUTPUT_TILE_X * OUTPUT_TILE_Y * OUTPUT_TILE_Z * sizeof(PPMPixel));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // Copy filter to constant memory on device
    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter_h, sizeof(Filter3D));
    hipDeviceSynchronize();

    // Set grid and block dims for tiled input/outputs
    dim3 dim_block(BLOCK_SIZE + FILTER_SIZE / 2, BLOCK_SIZE + FILTER_SIZE / 2, INPUT_TILE_Z);
    dim3 dim_grid((INPUT_TILE_X + 1) / BLOCK_SIZE + 1,
                  (INPUT_TILE_Y + 1) / BLOCK_SIZE + 1,
                  1);

    // Loop over x, y, z dimensions of the video and for each chunk process it with a kernel call
    for (int z = 0; z < totalFrames; z+=OUTPUT_TILE_Z)
    {
        // For each z, load INPUT_TILE_Z frames from disk
        loadFrames(inputFrames, z, totalFrames);
        for (int y = 0; y < image->y; y+=OUTPUT_TILE_Y)
        {
            for (int x = 0; x < image->x; x+=OUTPUT_TILE_X)
            {
                // For each x y z get a chunk of pixels from the input frames
                getPixels(inputFrames, inputData_h, x, y, z, image->x, image->y, totalFrames);
                // Copy pixels to device memory
                hipMemcpy(imageData_d, inputData_h, INPUT_TILE_X * INPUT_TILE_Y * INPUT_TILE_Z * sizeof(PPMPixel),
                           hipMemcpyHostToDevice);
                begin = clock();
                // Process pixels with convolution kernel
                convolution<<<dim_grid, dim_block>>>(imageData_d, outputData_d);
                cuda_ret = hipDeviceSynchronize();
                if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

                end = clock();
                time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
                // Copy processed pixels from device to host
                hipMemcpy(outputData_h, outputData_d, OUTPUT_TILE_X * OUTPUT_TILE_Y * OUTPUT_TILE_Z * sizeof(PPMPixel),
                           hipMemcpyDeviceToHost);
                // Write pixels to outputFrames in memory
                writePixels(outputData_h, outputFrames, x, y, z, image->x, image->y);
            }
        }
        // Write output frames to disk
        writeFrames(outputFrames, z, totalFrames);

    }

    // Free device and host memory
    free(inputData_h);
    free(outputData_h);
    for (int i = 0; i < INPUT_TILE_Z; i++)
        free(inputFrames[i].data);
    for (int i = 0; i < OUTPUT_TILE_Z; i++)
        free(outputFrames[i].data);
    hipFree(imageData_d);
    hipFree(outputData_d);

    // Combine frames into a single video with ffmpeg
    if (!system(NULL)) { exit (EXIT_FAILURE);}
    sprintf(ffmpegString, "ffmpeg -framerate 24 -i ../outfiles/tmp%%03d.ppm -c:v libx264 -r 30 -pix_fmt yuv420p ../outfilter.mp4");
    system (ffmpegString);

    printf("%f seconds spent\n", time_spent);

}
